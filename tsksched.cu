#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;
//You can also find min using atomicMin or using reduction
__device__ int findMin(int m, int* gpuCounter)
{   
    int min = gpuCounter[0];
    int index = 0;
    for(int i = 1; i < m; i++)
    {
        if(gpuCounter[i] < min)
        {  
            min = gpuCounter[i];
            index = i;
        }
    }
    return index;
}

__global__ void dkernel (int m, int n, int* gpuExecutionTime, int* gpuPriority, int* gpuFirst, int* gpuCounter, int* globalCounter, int* gpuResult)
{
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    
     while (*globalCounter < n)
        {  
            if(id < m)
            {
        
            int globalCounterTemp = *globalCounter;
            int taskCoreNo;

            if (gpuPriority[globalCounterTemp] != gpuPriority[globalCounterTemp-1])
            {
                if (globalCounterTemp != 0)
                    {
                        int prev = gpuFirst[gpuPriority[globalCounterTemp - 1]]; 
                        if (gpuCounter[id] < gpuCounter[prev] - gpuExecutionTime[globalCounterTemp - 1])
                        {
                            gpuCounter[id] = gpuCounter[prev] - gpuExecutionTime[globalCounterTemp - 1];
                        }
                    }
            }

            //Checking if it is the first task of particular priority
            if (gpuFirst[gpuPriority[globalCounterTemp]] == -1)
            {
                //If not: then find free core
                              
                taskCoreNo = findMin(m, gpuCounter);
              
                gpuFirst[gpuPriority[globalCounterTemp]] = taskCoreNo;

            }
            else
            {
                //Aleady core number is known
                taskCoreNo =  gpuFirst[gpuPriority[globalCounterTemp]];
            }

            //If core on which task should run is same as threadID
            if(taskCoreNo == id)
            {

                if (globalCounterTemp == 0)
                {
                    //If it is a very very first task OR Task 0
                    gpuCounter[id] += gpuExecutionTime[globalCounterTemp]; 

                    gpuResult[globalCounterTemp] = gpuCounter[id];
                }
                else
                {
                    //Check if its priority is same as previous task
                    if (gpuPriority[globalCounterTemp] == gpuPriority[globalCounterTemp-1])
                    {
                        gpuCounter[id] += gpuExecutionTime[globalCounterTemp]; 
                        gpuResult[globalCounterTemp] = gpuCounter[id];
                    }
                    else
                    {
                            gpuCounter[id] += gpuExecutionTime[globalCounterTemp];
                            gpuResult[globalCounterTemp] = gpuCounter[id];                            
                        
                    }

                }

                //Go for next task in ready queue
                *globalCounter = *globalCounter + 1;
            }
            //Else do nothing


        }
            //Add a barrier here for synchronization
        __syncthreads();

    }

    // printf ("\nReached here %d", id);
}


//Complete the following function
void operations ( int m, int n, int *executionTime, int *priority, int *result )  {
	// Allocate Cuda memory, copy from host into cuda memory 
	
	// call the kernels for doing required computations
	
	// copy the result back
	
	//Initialization : This can be parallelized easily
	 int *first = (int *) malloc ( (m) * sizeof (int) );
    for ( int i=0; i<m; i++ )  {
        first[i] = -1;
    }
    
    int *counter = (int *) malloc ( (m) * sizeof (int) );
    for ( int i=0; i<m; i++ )  {
        counter[i] = 0;
    }
	
	 int *gpuResult;
    hipMalloc( &gpuResult, sizeof(int) * (n) );
	hipMemcpy(gpuResult, result, sizeof(int) * (n), hipMemcpyHostToDevice);
	
    
    int *gpuFirst;
    hipMalloc( &gpuFirst, sizeof(int) * (m) );
	hipMemcpy(gpuFirst, first, sizeof(int) * (m), hipMemcpyHostToDevice);

    
    int *gpuCounter;
    hipMalloc( &gpuCounter, sizeof(int) * (m) );
	hipMemcpy(gpuCounter, counter, sizeof(int) * (m), hipMemcpyHostToDevice);

    int* globalCounter;
    int *initCounter = 0;
    hipMalloc(&globalCounter, sizeof(int));
    hipMemcpy(globalCounter, initCounter, sizeof(int), hipMemcpyHostToDevice);


    int *gpuExecutionTime;
    hipMalloc( &gpuExecutionTime, sizeof(int) * (n) );
	hipMemcpy(gpuExecutionTime, executionTime, sizeof(int) * (n), hipMemcpyHostToDevice);

    int *gpuPriority;
    hipMalloc( &gpuPriority, sizeof(int) * (n) );
	hipMemcpy(gpuPriority, priority, sizeof(int) * (n), hipMemcpyHostToDevice);

	
    dkernel <<<1, m>>> (m, n, gpuExecutionTime, gpuPriority, gpuFirst, gpuCounter, globalCounter, gpuResult);
    hipDeviceSynchronize();

	hipMemcpy(result, gpuResult, n * sizeof(int), hipMemcpyDeviceToHost);  
	

}

int main(int argc,char **argv)
{
    int m,n;
    //Input file pointer declaration
    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");
    
    //Checking if file ptr is NULL
    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &m );      //scaning for number of cores
    fscanf( inputfilepointer, "%d", &n );      //scaning for number of tasks
   
   //Taking execution time and priorities as input	
    int *executionTime = (int *) malloc ( n * sizeof (int) );
    int *priority = (int *) malloc ( n * sizeof (int) );
    for ( int i=0; i< n; i++ )  {
            fscanf( inputfilepointer, "%d", &executionTime[i] );
    }

    for ( int i=0; i< n; i++ )  {
            fscanf( inputfilepointer, "%d", &priority[i] );
    }

    //Allocate memory for final result output 
    int *result = (int *) malloc ( (n) * sizeof (int) );
    for ( int i=0; i<n; i++ )  {
        result[i] = 0;
    }
    
     hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    hipEventRecord(start,0);

    //==========================================================================================================
	
	operations ( m, n, executionTime, priority, result ); 
	
    //===========================================================================================================
    
    
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken by function to execute is: %.6f ms\n", milliseconds);
    
    // Output file pointer declaration
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    //Total time of each task: Final Result
    for ( int i=0; i<n; i++ )  {
        fprintf( outputfilepointer, "%d ", result[i]);
    }

    fclose( outputfilepointer );
    fclose( inputfilepointer );
    
    free(executionTime);
    free(priority);
    free(result);
    
    
    
}
